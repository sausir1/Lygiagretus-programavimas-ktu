#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
#include ""
#include <fstream>
#include <string>
//L!YD>Vgb8


using namespace std;

using namespace std;

auto const DATASIZE = 26;
const int gpu_threads = 10;

class Company
{
public:
    string name;
    int employees;
    double avgSalary;

    Company() {}
    Company(string name, int emp, double avg)
    {
        this->name = name;
        this->employees = emp;
        this->avgSalary = avg;
    }

};
class CompanyWithComputedValue
{
public:
    Company company;
    double root;

    CompanyWithComputedValue(string name, int employees, double salaries, double rootas)
    {
        company.name = name;
        company.employees = employees;
        company.avgSalary = salaries;
        root = rootas;
    }

    CompanyWithComputedValue() {}

};
void ReadFile(string file, Company companies[])
{
    int index = 0;
    ifstream input(file);
    int k = 0;
    string name;
    int empl;
    double avg;
    while (input)
    {
        k++;
        string line;
        getline(input, line);
        if (k == 1) { name = line; k++; }
        if (k == 3) { empl = stoi(line) ; k++; };
        if (k == 5)
        {
            avg = atof(line.c_str());
            companies[index].name = name;
            companies[index].employees = empl;
            companies[index].avgSalary = avg;
            k = 0;
            index++;
        }
    }
    input.close();
}

__global__ void run_on_gpu();
__device__ void execute(const char* name);
int main() {

    Company c[DATASIZE];
    string file = "";
    int choice = 0;
    cout << "Kuri faila duomenu faila naudoti?" << endl;
    cin >> choice;
    cout << "Choice =" << choice << endl;
    if(choice == 1){
        file ="IFF87_SirvydasS_L2_dat1.txt";
    }
    if(choice== 2){
        file= "IFF87_SirvydasS_L2_dat2.txt";
    }
    if(choice == 3){
        file = "IFF87_SirvydasS_L2_dat3.txt";
    }
    cout << "pasirinktas failas: " << file << endl;
    ReadFile(file,c);
    cout << "Nuskaityta" << endl;
    cout << c[0].name << endl;
    int* postproc = 0;
    run_on_gpu<<<1, gpu_threads>>>(postproc);
    hipDeviceSynchronize();
    cout << postproc << endl;
}


__global__ void run_on_gpu(int* kiek) {
    int chunkSize=DATASIZE/gpu_threads;
    int to_process =0;
    int thread_id = threadIdx.x;
    
    if((DATASIZE % gpu_threads) != 0)
    {
        to_process = (thread_id == gpu_threads - 1? chunkSize : chunkSize + 1);
        printf("Darbine gija Nr. %d to_process=%d\n",thread_id,to_process);
    }
    kiek +=to_process;
}

__device__ void execute(const char* name) {
    printf("%s: first\n", name);
    printf("%s: second\n", name);
    printf("%s: third\n", name);
}
